#include "hip/hip_runtime.h"
#include "cudapfem2particlehandler.cuh"

#include <deal.II/base/timer.h>

#include "../pfem2solver.h"
#include "cuda_helper.cuh"
#include "cudapfem2fem.cuh"
#include "cudapfem2mapping.cuh"
#include "cudapfem2finiteelement.cuh"

template<int dim>
__global__ void kernelSeedParticlesIntoCell(cudaPfem2Particle<dim> *particles, const cudaPfem2Cell<dim> *cells,
	const unsigned int *cellPartsIndices, int n)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n){
		const cudaPfem2Cell<dim> *cell = cells + i;

		double h[dim];
		for(int j = 0; j < dim; ++j)
			h[j] = 1.0 / d_quantities[j];

		cudaPfem2Particle<dim> *cellParticlesStart = particles + i * d_cellPartsCount;
		int particleID = i * d_cellPartsCount;

		double localPosition[dim], globalPosition[dim];

		for(int j = 0; j < d_cellPartsCount; ++j){
			for(int k = 0; k < dim; ++k)
				localPosition[k] = (cellPartsIndices[k] + 0.5) * h[k];

			cudaPfem2Mapping::transform_local_to_global<dim>((double*)&globalPosition, localPosition, cell->get_vertex_coords());
			cudaPfem2Particle<dim> particle((double*)&globalPosition, (double*)&localPosition, particleID++);
			particle.cell = cell;

			*(cellParticlesStart + j) = particle;
		}
	}
}

template<int dim>
__global__ void kernelCorrectParticleVelocity(cudaPfem2Particle<dim> *particles, const double *solutionV, const double *oldSolutionV, int n)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n){
		cudaPfem2Particle<dim> *particle = particles + i;

		double deltaV[dim]{0.0};
		double shapeValue;
		types::global_dof_index jDofIndex;

		for(int j = 0; j < GeometryInfo<dim>::vertices_per_cell; ++j){
			shapeValue = cudaPfem2FiniteElement::shape_value<dim>(j, particle->reference_location);
			jDofIndex = particle->cell->get_dof_indices()[j];

			for(int k = 0; k < dim; ++k)
				deltaV[k] += shapeValue * (solutionV[jDofIndex + k * d_ndofs] - ((oldSolutionV) ? oldSolutionV[jDofIndex + k * d_ndofs] : 0.0));
		}

		for(int k = 0; k < dim; ++k)
			particle->velocity[k] += deltaV[k];
	}
}

template<int dim>
__global__ void kernelTransferParticles(cudaPfem2Particle<dim> *particles, const double *solutionV, int n)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n){
		cudaPfem2Particle<dim> *particle = particles + i;

		double vel[dim]{0.0};
		double shapeValue;
		types::global_dof_index jDofIndex;

		for(int j = 0; j < GeometryInfo<dim>::vertices_per_cell; ++j){
			shapeValue = cudaPfem2FiniteElement::shape_value<dim>(j, particle->reference_location);
			jDofIndex = particle->cell->get_dof_indices()[j];

			for(int k = 0; k < dim; ++k)
				vel[k] += shapeValue * solutionV[jDofIndex + k * d_ndofs];
		}

		for(int k = 0; k < dim; ++k){
			particle->location[k] += d_timestep * vel[k];
			particle->velocity_ext[k] = vel[k];
		}
	}
}

template<int dim>
__global__ void kernelPrepareProjection(double *projectedVelocity, double *projectedWeights, const double value, int n)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n){
		for(int k = 0; k < dim; ++k)
			projectedVelocity[i + k * d_ndofs] = value;

		projectedWeights[i] = value;
	}
}

template<int dim>
__global__ void kernelProjectParticleVelocity(double *projectedVelocity, double *projectedWeights, const cudaPfem2Particle<dim> *particles, int n)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n){
		const cudaPfem2Particle<dim> *particle = particles + i;

		double shapeValue;
		types::global_dof_index jDofIndex;

		for(int j = 0; j < GeometryInfo<dim>::vertices_per_cell; ++j){
			shapeValue = cudaPfem2FiniteElement::shape_value<dim>(j, particle->reference_location);
			jDofIndex = particle->cell->get_dof_indices()[j];

			for(int k = 0; k < dim; ++k)
				atomicAdd(&projectedVelocity[jDofIndex + k * d_ndofs], shapeValue * particle->velocity[k]);

			atomicAdd(&projectedWeights[jDofIndex], shapeValue);
		}
	}
}

template<int dim>
__global__ void kernelUpdateNodeVelocity(double *solutionV, const double *projectedVelocity, const double *projectedWeights, int n)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n){
		unsigned int dofNumber;
		for(int k = 0; k < dim; ++k){
			dofNumber = i + k * d_ndofs;
			solutionV[dofNumber] = projectedVelocity[dofNumber] / projectedWeights[i];
		}
	}
}

template<int dim>
cudaPfem2ParticleHandler<dim>::cudaPfem2ParticleHandler(const FE_Q<dim> *finite_element)
	: pfem2ParticleHandler<dim>(finite_element)
{

}

template<int dim>
cudaPfem2ParticleHandler<dim>::~cudaPfem2ParticleHandler()
{
	hipFree(d_particles);
	hipFree(d_projectedVelocity);
	hipFree(d_projectedWeights);
	hipFree(d_quantities);
	hipFree(d_cellPartsIndices);
}

template<int dim>
void cudaPfem2ParticleHandler<dim>::seed_particles()
{
	pfem2ParticleHandler<dim>::seed_particles();

    //generate possible combinations of indices for cell parts within a single cell
	const unsigned int cellPartsCount = this->fill_cell_parts_indices();

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_quantities), &this->quantities, sizeof(unsigned int) * dim, 0, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cellPartsCount), &cellPartsCount, sizeof(unsigned int), 0, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc(&d_cellPartsIndices, sizeof(unsigned int) * dim * cellPartsCount));
	checkCudaErrors(hipMemcpy(d_cellPartsIndices, this->cellPartsIndices.data(), sizeof(unsigned int) * dim * cellPartsCount, hipMemcpyHostToDevice));

	int n_cells = this->mainSolver->getTriangulation().n_cells();
	currentParticleCount = cellPartsCount * n_cells;
	checkCudaErrors(hipMalloc(&d_particles, sizeof(cudaPfem2Particle<dim>) * currentParticleCount));

	unsigned int blocks = blocksForSize(n_cells);

	this->femSolver = static_cast<cudaPfem2Fem<dim>*>(&this->mainSolver->getFemSolver());
	const unsigned int n_dofs = femSolver->getDoFhandler().n_dofs();
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_ndofs), &n_dofs, sizeof(unsigned int), 0, hipMemcpyHostToDevice));
	kernelSeedParticlesIntoCell<dim><<<blocks, gpuThreads>>>(d_particles, femSolver->getCells(), d_cellPartsIndices, n_cells);
	getLastCudaError("Particle seeding");

	//call correct_particle_velocity() to initialize the particle velocity field
	blocks = blocksForSize(currentParticleCount);
	kernelCorrectParticleVelocity<dim><<<blocks, gpuThreads>>>(d_particles, femSolver->getDeviceSolutionV(), nullptr, currentParticleCount);
	getLastCudaError("Particle velocity field initialization");

	const double time_step = this->mainSolver->getParameterHandler().getTimeStep();
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_timestep), &time_step, sizeof(double), 0, hipMemcpyHostToDevice));

	//prepare vectors for particle velocity projection
	checkCudaErrors(hipMalloc(&d_projectedVelocity, sizeof(double) * n_dofs * dim));
	checkCudaErrors(hipMalloc(&d_projectedWeights, sizeof(double) * n_dofs));
}

template <int dim>
void cudaPfem2ParticleHandler<dim>::correct_particle_velocity()
{
	pfem2ParticleHandler<dim>::correct_particle_velocity();

	unsigned int blocks = blocksForSize(currentParticleCount);
	kernelCorrectParticleVelocity<dim><<<blocks, gpuThreads>>>(d_particles, femSolver->getDeviceSolutionV(), femSolver->getDeviceOldSolutionV(), currentParticleCount);
	getLastCudaError("Particle velocity correction");
}

template <int dim>
void cudaPfem2ParticleHandler<dim>::project_particle_fields()
{
	const unsigned int n_dofs = this->femSolver->getDoFhandler().n_dofs();

	//set the projected velocity and weights vectors to zero
	unsigned int blocks = blocksForSize(n_dofs);
	kernelPrepareProjection<dim><<<blocks, gpuThreads>>>(d_projectedVelocity, d_projectedWeights, 0.0, n_dofs);
	getLastCudaError("Projection vectors preparation");

	//project the particle velocity to intermediate vectors
	blocks = blocksForSize(currentParticleCount);
	kernelProjectParticleVelocity<dim><<<blocks, gpuThreads>>>(d_projectedVelocity, d_projectedWeights, d_particles, currentParticleCount);
	getLastCudaError("Particle velocity projection");

	//update the node velocities
	blocks = blocksForSize(n_dofs);
	kernelUpdateNodeVelocity<dim><<<blocks, gpuThreads>>>(femSolver->getDeviceSolutionV(), d_projectedVelocity, d_projectedWeights, n_dofs);
	getLastCudaError("Node velocity update");

	pfem2ParticleHandler<dim>::project_particle_fields();
}
